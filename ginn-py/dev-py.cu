// Copyright 2022 Bloomberg Finance L.P.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <pybind11/pybind11.h>

#include <ginn/dev.h>

#include <ginn-py/dev-py.h>

namespace ginn {
namespace python {

namespace py = pybind11;

inline int gpus_() {
  int num_gpus = -1;
  try {
    GINN_CUDA_CALL(hipGetDeviceCount(&num_gpus));
  } catch (const CudaError&) { return 0; }
  return num_gpus;
}

void barrier() {
  // This is needed for proper timing since cuda calls are async and cpu code
  // continues execution immediately.
  for (int i = 0; i < gpus_(); i++) {
    GINN_CUDA_CALL(hipSetDevice(i));
    GINN_CUDA_CALL(hipDeviceSynchronize());
  }
}

void bind_dev_gpu(py::module_& m) {
  using namespace py::literals;

  py::class_<Device<GPU>, std::shared_ptr<Device<GPU>>>(m, "BaseGpuDevice")
      .def_property_readonly("kind", &Device<GPU>::kind)
      .def_property_readonly("id", &Device<GPU>::id)
      .def_property_readonly("precedence", &Device<GPU>::precedence);

  py::class_<GpuDevice, Device<GPU>, std::shared_ptr<GpuDevice>>(m,
                                                                 "GpuDevice");

  m.def("Gpu", &Gpu, py::arg("gpu_idx") = 0);
  m.def("gpu", &gpu, py::arg("gpu_idx") = 0);

  py::class_<PreallocGpuDevice,
             Device<GPU>,
             std::shared_ptr<PreallocGpuDevice>>(m, "PreallocGpuDevice")
      .def("clear", &PreallocGpuDevice::clear)
      .def_property_readonly("size", &PreallocGpuDevice::size)
      .def_property_readonly("used", &PreallocGpuDevice::used);

  m.def("PreallocGpu",
        py::overload_cast<size_t, size_t>(&PreallocGpu),
        "idx"_a,
        "size"_a);

  m.def("gpus", &gpus_);
  m.def("barrier", &barrier);
}

} // namespace python
} // namespace ginn
